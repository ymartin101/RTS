#include "hip/hip_runtime.h"

/* ****************** Device Code ******************

 * Geometry node programs
 	* Intersection
 		* After OptiX searches through acceleration structure, test the ray for intersection against appropriate triangles
 		* Determine which triangle is the actual intersected triangle
 		* Interpolate surface normals
 	* Bounding box
 		* Computes axis-aligned bounding boxes

 ************************************************ */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "ray_tracer.h"

using namespace optix;

/* Declare variables */

// Variables with attributes
rtDeclareVariable(double3, normal, attribute normal, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

// User input variables (variables passed from the host)
rtBuffer < uint3, 1 > dbuf_triangles;				// Target's triangle/vertex indices
rtBuffer < double3, 1 > dbuf_triVertices;			// Target's vertices
rtBuffer < double3, 1 > dbuf_normals;				// Target's vertex normals
rtDeclareVariable(bool, d_interpolate_smooth, , );	// Enable/disable interpolation (curved surfaces)


/* Device functions */

// Function to make double3 variable
__device__ double3 to_double3(double inx, double iny, double inz)
{
	double3 out;
	out.x = inx;
	out.y = iny;
	out.z = inz;
	return out;
}

// Function to convert float3 to double3
__device__ double3 float3_to_double3(float3 in)
{
	double3 out;
	out.x = in.x;
	out.y = in.y;
	out.z = in.z;
	return out;
}


// Function to subtract double3s
__device__ double3 operator-(double3 a, double3 b)
{
	return to_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

// Function to multiply double with double3
__device__ double3 operator*(double a, double3 b)
{
	return to_double3(a * b.x, a * b.y, a * b.z);
}

// Function to compute cross product of two double3s
__device__ double3 crossd3(double3 a, double3 b)
{
	return to_double3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

// Function to compute dot product of two double3s
__device__ double dotd3(double3 a, double3 b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;;
}

// Function to compute length product of double3 and return it as a double
__device__ double lengthd3(double3 in)
{
	return sqrt(in.x*in.x + in.y*in.y + in.z*in.z);
}

// Function to normalise double3 input
__device__ double3 normalised3(double3 in)
{
	double norm = lengthd3(in);
	return to_double3(in.x/norm, in.y/norm, in.z/norm);
}

// Function to return smallest input double
__device__ double fmind(double a, double b)
{
	return a < b ? a : b;
}

// Function to return smallest input double3
__device__ double3 fmind3(double3 a, double3 b)
{
	return to_double3(fmind(a.x, b.x), fmind(a.y, b.y), fmind(a.z, b.z));
}

// Function to return largest input double
__device__ double fmaxd(double a, double b)
{
	return a > b ? a : b;
}

// Function to return largest input double3
__device__ double3 fmaxd3(double3 a, double3 b)
{
	return to_double3(fmaxd(a.x, b.x), fmaxd(a.y, b.y), fmaxd(a.z, b.z));
}

/* Function to perform intersection tests for doubles and double3s; modified version of function in <optixu/optixu_math_namespace.h> */
__device__ bool intersect_triangle_doubles(const Ray& ray, const double3& p0, const double3& p1, const double3& p2, double3& n,
                                            double&  t, double&  beta, double&  gamma)
{
	const double3 e0 = p1 - p0;
	const double3 e1 = p0 - p2;
	n = crossd3(e1, e0);

	// Note: prevHitPoint here is the ray origin RELATIVE to the intersection point as the OVERALL ORIGIN, i.e. (this ray origin =  intersection point - actual ray origin)
	const double3 e2 = (1/dotd3(n, prd.rayDirection)) * (p0 - prd.prevHitPoint);
	const double3 i = crossd3(prd.rayDirection, e2);
	
	beta = dotd3(i, e1);
	gamma = dotd3(i, e0);
	t = dotd3(n, e2);

	return ( (t < ray.tmax) & (t > ray.tmin) & (beta >= 0.0f) & (gamma >= 0.0f) & (beta + gamma <= 1) );
}


/* Intersection Program */

RT_PROGRAM void intersect(int prim_index) // prim_index is computed by OptiX functions and acceleration structure
{
	rtPrintf("Entering Intersect with prim_index: %d!\n\n", prim_index);

	// Triangle's vertex indices
	unsigned int v_idx0 = dbuf_triangles[prim_index].x;
	unsigned int v_idx1 = dbuf_triangles[prim_index].y;
	unsigned int v_idx2 = dbuf_triangles[prim_index].z;

	// Vertex coordinates relative to the origin
	double3 p0 = dbuf_triVertices[v_idx0];
	double3 p1 = dbuf_triVertices[v_idx1];
	double3 p2 = dbuf_triVertices[v_idx2];

	// // Or this? Should be same...
	// double3 p0 = to_double3(dbuf_triVertices[v_idx0].x, dbuf_triVertices[v_idx0].y, dbuf_triVertices[v_idx0].z);
	// double3 p1 = to_double3(dbuf_triVertices[v_idx1].x, dbuf_triVertices[v_idx1].y, dbuf_triVertices[v_idx1].z);
	// double3 p2 = to_double3(dbuf_triVertices[v_idx2].x, dbuf_triVertices[v_idx2].y, dbuf_triVertices[v_idx2].z);

	// printf("double p0: %e, %e, %e\n", p0.x, p0.y, p0.z);
	
	// Compute ray-triangle intersection
	double3 n;
	double t, beta, gamma;
	if (intersect_triangle_doubles( ray, p0, p1 , p2, n, t, beta, gamma )){ // Modified version of function in <optixu/optixu_math_namespace.h>
		if (rtPotentialIntersection( t ) ) {								// This OptiX function, rtPotentialIntersection, must be used here

			// Find the associated vertex normals
			double3 n0 = dbuf_normals[v_idx0];
			double3 n1 = dbuf_normals[v_idx1];
			double3 n2 = dbuf_normals[v_idx2];

			// If interpolation is enabled
			if (d_interpolate_smooth == true) {

				// If number of vertex normals > number of vertices, assume rect shape; vert_normals (8 for rect) was set to face normals (12 for rect)
				if (dbuf_normals.size() > dbuf_triVertices.size()) {
					normal = dbuf_normals[prim_index];
				}
				else {	// Interpolate vertex normals
					normal = to_double3(n1.x*beta + n2.x*gamma + n0.x*(1.0f - beta - gamma),
										n1.y*beta + n2.y*gamma + n0.y*(1.0f - beta - gamma),
										n1.z*beta + n2.z*gamma + n0.z*(1.0f - beta - gamma));
				}

				// Normalise as a double3 to preserve precision
				normal = normalised3(normal);
			}

			else {
				// Use n by default; locally flat face normal
				normal = normalised3(n);
			}

			// OptiX reporting function; must be used here
			rtReportIntersection(0);
		}
	}
}

/* Bounding Box Program */

RT_PROGRAM void bound (int prim_index, float result[6]) // prim_index is computed by OptiX functions and acceleration structure
{
	rtPrintf("Entering Bound with prim_index: %d!\n\n", prim_index);

	// For the triangle index (prim_index), search the buffer for the associated vertex coordinates
	unsigned int v_idx0 = dbuf_triangles[prim_index].x;
	unsigned int v_idx1 = dbuf_triangles[prim_index].y;
	unsigned int v_idx2 = dbuf_triangles[prim_index].z;

	const double3 v0 = dbuf_triVertices[v_idx0];
	const double3 v1 = dbuf_triVertices[v_idx1];
	const double3 v2 = dbuf_triVertices[v_idx2];
	const double area = lengthd3(crossd3(v1 - v0, v2 - v0));

	// OptiX's utility class aabb ("axis aligned bounding box") is used to compute bounding boxes
	optix::Aabb* aabb = (optix::Aabb*)result;

	// Perform computation
	if (area > 0.0f && !isinf(area)) {							// "isinf" checks if value is infinite; unrelated to sine
		double3 d3min = fmind3( fmind3(v0, v1), v2 );
		double3 d3max = fmaxd3( fmaxd3(v0, v1), v2 );

		// Only use float right at the end; maximise precision; use CUDA function to convert double to float and round "outwards" to closest float;
		// Ensures that every ray that would have hit the "double-precision" geometry will still hit the "float-precision" bounding box
		aabb->m_min = make_float3(__double2float_rd(d3min.x), __double2float_rd(d3min.y), __double2float_rd(d3min.z));
		aabb->m_max = make_float3(__double2float_ru(d3max.x), __double2float_ru(d3max.y), __double2float_ru(d3max.z));
	}
	else
		aabb->invalidate();	// OptiX function to invalidate
}
