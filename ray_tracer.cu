#include "hip/hip_runtime.h"

/* ****************** Device Code ******************

 * Context node programs
 	* Ray generation
 		* Initialise PRD, output buffer
 		* Compute ray directions, rays
 		* Call rtTrace
 		* Save results to output buffer
 	* Miss
 		* Determines if the ray is valid
 		* Computes end points if applicable on the receiving array
 		* Computes reflection coefficient of ray
 	* Exception
 		* Can be used for various debugging

 ************************************************ */

#include <optix_world.h>
#include "ray_tracer.h"

using namespace optix;

/* Declare variables */

// Variables with attributes, defined for the first time here
rtDeclareVariable(uint3, launchIndex, rtLaunchIndex, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

// User input variables (variables passed from the host)
rtBuffer < PerRayData, 1 > dbuf_results;			// rtBuffer < Type, dim >
rtBuffer < double3, 1 > dbuf_sphCentre;		// RxSphere centres
rtBuffer < double, 1 > dbuf_sphRadius;		// RxSphere radii
rtBuffer < double, 1 > dbuf_minTheta;		// RxSphere minThetas
rtBuffer < double, 1 > dbuf_maxTheta;		// RxSphere maxThetas
rtBuffer < double, 1 > dbuf_minPhi;			// RxSphere minPhis
rtBuffer < double, 1 > dbuf_maxPhi;			// RxSphere maxPhis
rtDeclareVariable(rtObject, d_targets_all, , );
rtDeclareVariable(unsigned int, d_width, , );
rtDeclareVariable(double3, d_rayOrigin, , );
rtDeclareVariable(double3, d_txSpan, , );
rtDeclareVariable(double2, d_txDir, , );
rtDeclareVariable(unsigned int, d_rxsize, , );
rtDeclareVariable(unsigned int, d_maxRayTotal, , );
rtDeclareVariable(double, d_beamwidth_azi, , );
rtDeclareVariable(double, d_beamwidth_ele, , );


/* Device functions */

// Normalise an angle to the range (-M_PI to +M_PI)
__device__ void normalise_angle(double& angle) 
{
    while ( angle < -M_PI ) angle += 2*M_PI;
    while ( angle >  M_PI ) angle -= 2*M_PI;
}

// Check if a testAngle is between two other angles, a and b
__device__ bool angle_in_range(double testAngle, double a, double b)
{
    a -= testAngle;
    b -= testAngle;
    normalise_angle( a );
    normalise_angle( b );
    if ( a * b >= 0 )
    	return false;
    return fabs( a - b ) < M_PI;
}

// Function to make double3 variable
__device__ double3 to_double3(double inx, double iny, double inz)
{
	double3 out;
	out.x = inx;
	out.y = iny;
	out.z = inz;
	return out;
}

// Function to normalise double3 input
__device__ double3 normalised3(double3 in)
{
	double norm = sqrt(in.x*in.x + in.y*in.y + in.z*in.z);
	return to_double3(in.x/norm, in.y/norm, in.z/norm);
}

// Function to add double3s
__device__ double3 operator+(double3 a, double3 b)
{
	return to_double3(a.x + b.x, a.y + b.y, a.z + b.z);
}

// Function to subtract double3s
__device__ double3 operator-(double3 a, double3 b)
{
	return to_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

// Function to multiply double3 with double
__device__ double3 operator*(double3 a, double b)
{
	return to_double3(a.x * b, a.y * b, a.z * b);
}

// Function to compute cross product of two double3s
__device__ double3 crossd3(double3 a, double3 b)
{
	return to_double3(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

// Function to get the squared magnitude of a double3
__device__ double magsquared3(double3 a)
{
	return (a.x*a.x + a.y*a.y + a.z*a.z);
}

// Function to compute length product of double3 and return it as a double
__device__ double lengthd3(double3 in)
{
	return sqrt(in.x*in.x + in.y*in.y + in.z*in.z);
}

// Function to normalise float3 input
__device__ float3 normalise_float3(double in1, double in2, double in3)
{
	double norm = lengthd3(to_double3(in1, in2, in3));
	return make_float3(in1/norm, in2/norm, in3/norm);
}

// Function to convert double spherical coordinates to Cartesian coordinates (unit length)
__device__ double3 sph_to_cart(double azi, double ele)
{
	double3 cart;
	cart.x = cos(azi)*cos(ele);
    cart.y = sin(azi)*cos(ele);
    cart.z = sin(ele);
	return cart;
}


/* Ray Generation Program */

RT_PROGRAM void ray_generation()
{
	// Print statement for debugging; does not print unless enabled in host code
	rtPrintf("Entering ray generation program...\n\n");

	// Loop through target list, create grid nodes for each target, then send rays in the direction of the nodes
	unsigned int rayTotal = d_width*d_width*d_width;
	unsigned int rayIndex = launchIndex.z*d_width*d_width + launchIndex.y*d_width + launchIndex.x;		// Overall ray launch index

    // Find beam start-point relative to boresight centre; uses spherical coordinates (d_txDir: x = azimuth, y = elevation)
	// Uses d_txSpan: x = azimuth span, y = elevation span, z = launch range
	double3 beamStart = sph_to_cart(-d_txSpan.x/2, -d_txSpan.y/2);
	double3 beamEnd = sph_to_cart(d_txSpan.x/2, d_txSpan.y/2);

    // If only one ray is spawned, set the start position to the boresight/beam centre (in that dimension)
	double3 rayDir_d3;
    if (d_width == 1) {											// d_width = "d_height" = "d_depth"
    	rayDir_d3 = sph_to_cart(d_txDir.x, d_txDir.y);			// Unit length; azimuth and elevation are boresight direction
		// printf("[%.10e, %.10e, %.10e];\n", rayDir_d3.x, rayDir_d3.y, rayDir_d3.z);
	}
	else {

		// Compute ray direction using rectangular coordinates
		rayDir_d3.x = beamStart.x + (((beamEnd.x*(1 + d_txSpan.z)) - beamStart.x)/(d_width - 1)) * (launchIndex.x);	// d_txSpan.z is launch range
		rayDir_d3.y = beamStart.y + ((beamEnd.y - beamStart.y)/(d_width - 1)) * (launchIndex.y);
		rayDir_d3.z = beamStart.z + ((beamEnd.z - beamStart.z)/(d_width - 1)) * (launchIndex.z);
		rayDir_d3 = normalised3(rayDir_d3);

		// Form azimuth rotation matrix; NOTE: Right-hand rule applies along z-axis (like yaw)
		double Rot[3][3] = {{cos(d_txDir.x), -sin(d_txDir.x), 0}, \
							{sin(d_txDir.x), cos(d_txDir.x), 0}, \
							{0, 0, 1}};

		// Apply boresight azimuth rotation; matrix multiplication
		double3 rotated; rotated.x = 0; rotated.y = 0; rotated.z = 0;	// Set rotated variable
		rotated.x += Rot[0][0]*rayDir_d3.x + Rot[0][1]*rayDir_d3.y + Rot[0][2]*rayDir_d3.z;
		rotated.y += Rot[1][0]*rayDir_d3.x + Rot[1][1]*rayDir_d3.y + Rot[1][2]*rayDir_d3.z;
		rotated.z += Rot[2][0]*rayDir_d3.x + Rot[2][1]*rayDir_d3.y + Rot[2][2]*rayDir_d3.z;
		rayDir_d3 = normalised3(rotated);

		// Apply "y-axis" azimuth rotation; rotates y-axis by the same azimuth, then elevation rotation is applied using this "new y-axis"
		// All other terms of matrix multiplication is ZERO since x = z = 0; only leaves y-terms as y = 1
		rotated.x = 0; rotated.y = 0; rotated.z = 0;	// Reset rotated variable
		rotated.x += Rot[0][1];
		rotated.y += Rot[1][1];
		rotated.z += Rot[2][1];
		double3 orth_vec = normalised3(rotated);	// Rotated y-axis

		// Form elevation rotation matrix; see: https://stackoverflow.com/questions/6721544/circular-rotation-around-an-arbitrary-axis
		// NOTE: sin terms' signs are changed from usual formula; reverses elevation direction such that it agrees with the way used in RTS
		double Rot1[3][3] = {{cos(d_txDir.y) + orth_vec.x*orth_vec.x*(1 - cos(d_txDir.y)), orth_vec.x*orth_vec.y*(1 - cos(d_txDir.y)) + orth_vec.z*sin(d_txDir.y), orth_vec.x*orth_vec.z*(1 - cos(d_txDir.y)) - orth_vec.y*sin(d_txDir.y)}, \
							{orth_vec.y*orth_vec.x*(1 - cos(d_txDir.y)) - orth_vec.z*sin(d_txDir.y), cos(d_txDir.y) + orth_vec.y*orth_vec.y*(1 - cos(d_txDir.y)), orth_vec.y*orth_vec.z*(1 - cos(d_txDir.y)) + orth_vec.x*sin(d_txDir.y)}, \
							{orth_vec.z*orth_vec.x*(1 - cos(d_txDir.y)) + orth_vec.y*sin(d_txDir.y), orth_vec.z*orth_vec.y*(1 - cos(d_txDir.y)) - orth_vec.x*sin(d_txDir.y), cos(d_txDir.y) + orth_vec.z*orth_vec.z*(1 - cos(d_txDir.y))}};

		// Apply boresight elevation rotation; matrix multiplication
		rotated.x = 0; rotated.y = 0; rotated.z = 0;	// Reset rotated variable
		rotated.x += Rot1[0][0]*rayDir_d3.x + Rot1[0][1]*rayDir_d3.y + Rot1[0][2]*rayDir_d3.z;
		rotated.y += Rot1[1][0]*rayDir_d3.x + Rot1[1][1]*rayDir_d3.y + Rot1[1][2]*rayDir_d3.z;
		rotated.z += Rot1[2][0]*rayDir_d3.x + Rot1[2][1]*rayDir_d3.y + Rot1[2][2]*rayDir_d3.z;
		rayDir_d3 = rotated;
		// printf("[%.10e, %.10e, %.10e];\n", rayDir_d3.x, rayDir_d3.y, rayDir_d3.z);
	}

	// Spawn a ray; need direction and origin as float3s - possible loss in precision here
	float3 rayDir_f3 = normalise_float3(rayDir_d3.x, rayDir_d3.y, rayDir_d3.z);	// Normalise double3 and THEN convert to float3
	Ray ray = make_Ray(make_float3(d_rayOrigin.x, d_rayOrigin.y, d_rayOrigin.z), rayDir_f3, 0, SCENE_EPS, RT_DEFAULT_MAX);
	
	// Initialize PRD instance
	PerRayData prd;
	prd.reflDepth = 0;
	prd.refrDepth = 0;
	prd.maxRayIndex = 0;
	prd.rayLength = 0;
	prd.rayDirection = rayDir_d3;						// Used as double3 ray direction for precision
	prd.firstHitPoint = make_double3(0.f, 0.f, 0.f);
	prd.prevHitPoint = d_rayOrigin;						// Initially used as ray origin
	prd.refrIndex.x = 1; prd.refrIndex.y = 1;
	prd.power = 0;
	prd.doppler = 0;
	prd.received = -1;
	prd.end = false;

 	// Initialise output buffer
 	for (unsigned int i = 0; i < (d_maxRayTotal/rayTotal); i++) {	// In this file, rayTotal = rays transmitted; maxRayTotal includes all refractions
		dbuf_results[rayIndex + i*rayTotal].reflDepth = 0;
		dbuf_results[rayIndex + i*rayTotal].refrDepth = 0;
		dbuf_results[rayIndex + i*rayTotal].maxRayIndex = 0;
		dbuf_results[rayIndex + i*rayTotal].rayLength = 0;
		dbuf_results[rayIndex + i*rayTotal].rayDirection = make_double3(0, 0, 0);
		dbuf_results[rayIndex + i*rayTotal].firstHitPoint = make_double3(0, 0, 0);
		dbuf_results[rayIndex + i*rayTotal].prevHitPoint = make_double3(0, 0, 0);
		dbuf_results[rayIndex + i*rayTotal].refrIndex.x = 1; dbuf_results[rayIndex + i*rayTotal].refrIndex.y = 1;
		dbuf_results[rayIndex + i*rayTotal].power = 0;
		dbuf_results[rayIndex + i*rayTotal].doppler = 0;
		dbuf_results[rayIndex + i*rayTotal].received = -1;
		dbuf_results[rayIndex + i*rayTotal].end = false;
	}

	// Call OptiX's rtTrace ray traversal function
	rtTrace(d_targets_all, ray, prd);

	// When reflection rtTrace finishes ray traversal, save results; not all variables need to be saved
	dbuf_results[rayIndex].reflDepth = prd.reflDepth;
	dbuf_results[rayIndex].refrDepth = prd.refrDepth;
	dbuf_results[rayIndex].rayLength = prd.rayLength;
	dbuf_results[rayIndex].firstHitPoint = prd.firstHitPoint;
	dbuf_results[rayIndex].prevHitPoint = prd.prevHitPoint;
	dbuf_results[rayIndex].power = prd.power;
	dbuf_results[rayIndex].doppler = prd.doppler;
	dbuf_results[rayIndex].received = prd.received;
	// dbuf_results[rayIndex].rayDirection = prd.rayDirection;	// REMOVE
}


/* Miss Program */

RT_PROGRAM void miss()
{	
	// Print statement for debugging; does not print unless enabled in host code
 	rtPrintf("Entering miss!\n\n");

 	// If ray has not previously hit Earth
 	if (prd.end == false) {

		double A, B, C, discriminant;	// Set up variables for ray-sphere intersections
		double t[2] = {0, 0};

		// Iterate through every receiver and check if there is an intersection with ray
		for (unsigned int Rx_i = 0; Rx_i < d_rxsize; Rx_i++) {

			/* Compute ray-sphere (ray-receiver) intersection:
			Ray = o + td -> origin (x, y, z) + t.direction (x, y, z)
			Ray origin used as the prevHitPoint
			Sphere equation -> (x - cx)² + (y - cy)² + (z - cz)² - r² = 0
			Substitute x = x_Ray, y = y_Ray and z = z_Ray and get quadratic equation to solve for t
			(ox + tdx - cx)² + (oy + tdy - cy)² + (oz + tdz - cz)² - r² = 0
			t² (dx² + dy² + dz²) + \
			t (2(dx(ox - cx) + dy(oy - cy) + dz(oz - cz))) + \
			1 (ox² + oy² + oz² + cx² + cy² + cz² - 2(cx.ox + cy.oy + cz.oz) - r²) = 0
			--> At² + Bt + C = 0 */

			A = ((prd.rayDirection).x)*((prd.rayDirection).x) + ((prd.rayDirection).y)*((prd.rayDirection).y) + ((prd.rayDirection).z)*((prd.rayDirection).z);
			B = 2*((((prd.prevHitPoint).x - dbuf_sphCentre[Rx_i].x)*(prd.rayDirection).x) + \
				   (((prd.prevHitPoint).y - dbuf_sphCentre[Rx_i].y)*(prd.rayDirection).y) + \
				   (((prd.prevHitPoint).z - dbuf_sphCentre[Rx_i].z)*(prd.rayDirection).z));
			C = (prd.prevHitPoint).x*(prd.prevHitPoint).x + (prd.prevHitPoint).y*(prd.prevHitPoint).y + (prd.prevHitPoint).z*(prd.prevHitPoint).z + \
				(dbuf_sphCentre[Rx_i].x*dbuf_sphCentre[Rx_i].x) + \
				(dbuf_sphCentre[Rx_i].y*dbuf_sphCentre[Rx_i].y) + \
				(dbuf_sphCentre[Rx_i].z*dbuf_sphCentre[Rx_i].z) - \
				2*((dbuf_sphCentre[Rx_i].x*(prd.prevHitPoint).x) + (dbuf_sphCentre[Rx_i].y*(prd.prevHitPoint).y) + (dbuf_sphCentre[Rx_i].z*(prd.prevHitPoint).z)) - \
				dbuf_sphRadius[Rx_i]*dbuf_sphRadius[Rx_i];
			discriminant = B*B - 4*A*C;		// Discriminant

			// If roots of quadratic equation are real-valued, ray intersects sphere surface somewhere
			if (discriminant > 0.f) {

				// printf("Ray here!\n");

				// Solve for t (roots of quadratic equation)
				discriminant = sqrt(discriminant);
				t[0] = (-B - discriminant)/(2*A);	// First root
				t[1] = (-B + discriminant)/(2*A);	// Second root
				// printf("t[0]: %e, t[1]: %e\n", t[0], t[1]);

				// Roots loop
				unsigned int received_root = 2; 	// Default of 2; will be either 0 or 1 if ray hits receiver for this root
				for (int i = 0; i < 2; i++) {		// Use BOTH roots of quadratic equation in case there are two intersections

					// If t >= 0, root is valid; ignore for t < 0
					// Also, total ray length must be larger than incident scene epsilon; if not, assume error (monostatic "direct transmission")
					if ((t[i] >= 0) && ((prd.rayLength + t[i]) > SCENE_EPS) && ((prd.rayLength + t[i]) > SCENE_EPS_R)) {

						// printf("Here: %e, %e, %e\n", (prd.prevHitPoint).x, (prd.prevHitPoint).y, (prd.prevHitPoint).z);

						// End-point of a ray on the sphere's surface
						double3 endPoint;
						endPoint.x = (prd.prevHitPoint).x + t[i]*(prd.rayDirection).x;
						endPoint.y = (prd.prevHitPoint).y + t[i]*(prd.rayDirection).y;
						endPoint.z = (prd.prevHitPoint).z + t[i]*(prd.rayDirection).z;

						// Compute spherical angles; use as double for comparison with minima and maxima later
						// Get theta, phi relative to sphere centre; elevation always between -Pi/2 and Pi/2 since "r" uses +sqrt(...)
						double theta = atan2f((endPoint.y - dbuf_sphCentre[Rx_i].y), (endPoint.x - dbuf_sphCentre[Rx_i].x));	// Uses azimuth angle measured from x-axis towards ray vector component in xy-plane
						double phi = atan2f(endPoint.z - dbuf_sphCentre[Rx_i].z, sqrt(((endPoint.y - dbuf_sphCentre[Rx_i].y) * \
											(endPoint.y - dbuf_sphCentre[Rx_i].y)) + ((endPoint.x - dbuf_sphCentre[Rx_i].x) * \
											(endPoint.x - dbuf_sphCentre[Rx_i].x))));	// Uses elevation angle measured from xy-plane towards ray vector

						// Check that phi is within -M_PI/2 and +M_PI/2
			            if ((phi < -M_PI/2)){						// If phi is below -90 deg
			                theta += M_PI;							// Azimuth change
							phi = -M_PI - phi;						// Elevation change (e.g. to -180 - (-95) = -85 deg)
						}

			            if ((phi > M_PI/2)){						// If phi is above +90 deg
			                theta += M_PI;							// Azimuth change
							phi = M_PI - phi;						// Elevation change (e.g. from 180 - 95 = 85 deg)
						}

			            // Set up variables for angle comparisons
			            double d_maxTheta1 = dbuf_maxTheta[Rx_i];
			            double d_minTheta1 = dbuf_minTheta[Rx_i];
			            double d_maxTheta2 = d_maxTheta1;			// Copy of maxTheta1
		            	double d_minTheta2 = d_minTheta1;			// Copy of minTheta1
		            	double d_maxPhi1 = dbuf_maxPhi[Rx_i];
			            double d_minPhi1 = dbuf_minPhi[Rx_i];
			            double d_maxPhi2 = d_maxPhi1;				// Copy of maxPhi1
		            	double d_minPhi2 = d_minPhi1;				// Copy of minPhi1

			            // Check min. and max. bin Phis are within range of -M_PI/2 < Phi < M_PI/2; if not, create second azimuth/elevation regions
			            // Do not need to check maxPhi1 < -M_PI/2 or minPhi1 > M_PI/2; minimum maxPhi1 is -M_PI/2; maximum minPhi1 is M_PI/2
			            if ((d_minPhi1 < -M_PI/2)){					// If minPhi is below -M_PI/2  (e.g. minPhi1 = -95 deg, maxPhi1 = 55 deg)
			                d_maxTheta2 += M_PI;					// Second azimuth region max
							d_minTheta2 += M_PI;					// Second azimuth region min
							d_maxPhi2 = -M_PI - d_minPhi1;			// Second elevation region max (e.g. to -180 - (-95) = -85 deg)
							d_minPhi2 = -M_PI/2;					// Second elevation region min (e.g. from -90 deg)
							d_minPhi1 = -M_PI/2;					// First elevation region min (e.g. from -90 deg); max stays the same (e.g. 55 deg)
						}

			            if ((d_maxPhi1 > M_PI/2)){					// If maxPhi is above +M_PI/2 (e.g. maxPhi1 = 95 deg, minPhi1 = 55 deg)
			                d_maxTheta2 += M_PI;					// Second azimuth region max
							d_minTheta2 += M_PI;					// Second azimuth region min
							d_minPhi2 = M_PI - d_maxPhi1;			// Second elevation region min (e.g. from 180 - 95 = 85 deg)
							d_maxPhi2 = M_PI/2;						// Second elevation region max (e.g. to +90 deg)
							d_maxPhi1 = M_PI/2;						// First elevation region max (e.g. to +90 deg); min stays the same (e.g. 55 deg)
						}

						// Check if the ray hits the part of the Rx sphere we are observing (i.e. the "antenna")
						// MUST test theta/phi TOGETHER; ray could be within an theta range but not the ASSOCIATED phi range --> INVALID RECEIVAL
						// The below IF statements check if angles are INSIDE the angle ranges
						// For only one azimuth/elevation range, the OR below is useless; performs the same check twice since angle2s = angle1s
						if (((angle_in_range(theta, d_minTheta1, d_maxTheta1)) && (angle_in_range(phi, d_minPhi1, d_maxPhi1))) || \
							(angle_in_range(theta, d_minTheta2, d_maxTheta2)) && (angle_in_range(phi, d_minPhi2, d_maxPhi2)))
						{
							// Update received_root as this ray will be captured for this root i
							if (received_root == 2)				// For first root, default is 2; for second root, default is still 2 if first root not captured
								received_root = i;				// Becomes 0 (first root) or 1 (second root) if previous received_root is 2
							else if (t[received_root] > t[i])	// If previous t[received_root] is larger than current t[i], use current t[i] as new t[received_root]
								received_root = i;				// Use shortest path length; only used when capture occurs TWICE (both roots)
							
							// AFTER roots loop, the new received_root will be used for the recorded computations (see below)
							// If both roots worked, the smaller t[i] is used (first capture)
							// If the received_root = 0 is captured and received_root = 1 is not, received_root = 0 is used
							// If the received_root = 1 is captured and received_root = 0 is not, received_root = 1 is used
							// If neither root makes it this far, received_root = 2 is used (nothing is recorded)
						}
					}
				}

				/// AFTER roots loop is done, check if receiver was intersected for EITHER loop
				if (received_root < 2)	{	// At least one root resulted in a ray capture at the receiver

					// Ray terminated after capture
					prd.end = true;

					// Set i to received_root
					unsigned int i = received_root;

					// End-point of ray on the sphere's surface
					double3 endPoint;
					endPoint.x = (prd.prevHitPoint).x + t[i]*(prd.rayDirection).x;
					endPoint.y = (prd.prevHitPoint).y + t[i]*(prd.rayDirection).y;
					endPoint.z = (prd.prevHitPoint).z + t[i]*(prd.rayDirection).z;
					// printf("[%e, %e, %e];\n", endPoint.x, endPoint.y, endPoint.z);

					// Calculate (part of) the received power (narrow-band bistatic radar equation) and Doppler
					double3 RxRange;
					if ((prd.reflDepth == 0) && (prd.refrDepth == 0)) {				// If direct transmission
						RxRange = endPoint - d_rayOrigin;							// Range from Tx to Rx
						if (lengthd3(RxRange) >= SCENE_EPS) {						// Must be larger than scene epsilon
							prd.power = 1/(4*M_PI*4*M_PI*(magsquared3(RxRange)));	// Set power from Tx to Rx
							prd.doppler = 0;										// Zero relative motion --> zero Doppler
							prd.rayLength += t[i];									// Add rayLength
							prd.received = Rx_i;									// Ray has hit a receiver (index Rx_i)
						}
					}
					else {															// If not direct transmission
						RxRange = endPoint - prd.prevHitPoint;						// From last-hit target to Rx
						if (lengthd3(RxRange) >= SCENE_EPS_R) {						// Must be larger than scene epsilon (generally use SCENE_EPS_R here)
							prd.power *= 1/((magsquared3(RxRange))*4*M_PI*4*M_PI);	// Update power with RxRange and Rx (4*pi)
							prd.rayLength += t[i];									// Add rayLength
							prd.received = Rx_i;									// Ray has hit a receiver (index Rx_i)
						}
					}
				}
			}
		}
	}

	/// Scenario when ray hits the Earth (modelled as a sphere)

	// Print statement for debugging; does not print unless enabled in host code
	rtPrintf("This ray had at least one intersection and then entered miss!\n\n");

	// If the ray hits the Earth (or hits a target after reaching its maximum reflections and refractions) before hitting the receiver
	if (prd.end == false) {

		/* Compute ray-Earth (ray-receiver) intersection (same as for Rx sphere)
		Ray = o + td -> origin (x, y, z) + t.direction (x, y, z)
		Sphere equation -> x² + y² + z² - r² = 0; substitute x = x_Ray, y = y_Ray and z = z_Ray and get quadratic equation to solve for t
		(ox + tdx)² + (oy + tdy)² + (oz + tdz)² - r² = 0
		t² (dx² + dy² + dz²) + t (2(ox.dx + oy.dy + oz.dz) + (ox² + oy² + oz² - r²) = 0 --> At² + Bt + C = 0
		Ray overall origin = prd.prevHitPoint -> relative to overall coordinate system's origin */

		double d_earthRadius = 6378136;	// Earth radius [m]
		double A = ((prd.rayDirection).x)*((prd.rayDirection).x) + ((prd.rayDirection).y)*((prd.rayDirection).y) + ((prd.rayDirection).z)*((prd.rayDirection).z);
		double B = 2*((prd.prevHitPoint).x*(prd.rayDirection).x + (prd.prevHitPoint).y*(prd.rayDirection).y + (prd.prevHitPoint).z*(prd.rayDirection).z);
		double C = (prd.prevHitPoint).x*(prd.prevHitPoint).x + (prd.prevHitPoint).y*(prd.prevHitPoint).y + (prd.prevHitPoint).z*(prd.prevHitPoint).z - d_earthRadius*d_earthRadius;
		double discriminant = B*B - 4*A*C;	// Discriminant
		double t[2] = {0, 0};

		// If roots of quadratic equation are real-valued, ray intersects sphere surface somewhere
		if (discriminant > 0.f) {

			// Solve for t (roots of quadratic equation)
			discriminant = sqrt(discriminant);
			t[0] = (-B - discriminant)/(2*A);	// First root
			t[1] = (-B + discriminant)/(2*A);	// Second root

			for (int i = 0; i < 2; i++) {	// Use both roots of quadratic equation in case there are two intersections

				if ((t[i] >= 0) && (prd.rayLength > 0)) {	// If t >= 0, root is valid

					// Terminate the ray after this; ray has hit the Earth
					prd.end = true;

					// Update PRD
					prd.rayLength += t[i];				// Add rayLength

					// Print statement for debugging; may print twice if there are TWO Earth intersections
					// printf("Ray hit the Earth; rayLength = %e\n", prd.rayLength);
				}
			}
		}
	}
}

// // Exception Program
// RT_PROGRAM void exception() {
// 		rtPrintf("Entering exception program!\n\n");
// 		prd.rayLength = -1.f;
// 		const unsigned int code = rtGetExceptionCode();
// 		rtPrintf( "Caught exception 0x%X at launch index (%d, %d)\n", code, launchIndex.x, launchIndex.y );
// }
