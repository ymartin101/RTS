#include "hip/hip_runtime.h"

/* ****************** Device Code ******************

* Material node program
* Closest hit
	* After testing the ray for intersection against appropriate triangles
	* Compute hit-points, reflCoeffs, etc
	* Update PRD for intersecting ray
	* Re-launch ray in refraction/reflection direction

************************************************ */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "ray_tracer.h"

using namespace optix;

/* Declare variables */

// Variables with attributes
rtDeclareVariable(float, hit_t, rtIntersectionDistance, );
rtDeclareVariable(uint3, launchIndex, rtLaunchIndex, );
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(double3, normal, attribute normal, );

// User input variables (variables passed from the host)
rtBuffer < PerRayData, 1 > dbuf_results;			// rtBuffer < Type, dim >
rtBuffer < int, 2 > dbuf_targ_intersect;			// rtBuffer < Type, dim >
rtBuffer < double2, 2 > dbuf_rcs_angle;				// rtBuffer < Type, dim >
rtBuffer < double3, 1 > dbuf_targ_vel;				// Target positions
rtDeclareVariable(double3, d_rayOrigin, , );
rtDeclareVariable(unsigned int, d_maxReflDepth, , );
rtDeclareVariable(unsigned int, d_maxRefrDepth, , );
rtDeclareVariable(rtObject, d_targets_all, , );
rtDeclareVariable(double, d_targReflCoeff, , );
rtDeclareVariable(double, d_targRefrIndex, , );
rtDeclareVariable(unsigned int, d_targIndex, , );
rtDeclareVariable(unsigned int, d_width, , );


/* Device functions */

// Function to make double3 variable
__device__ double3 to_double3(double inx, double iny, double inz)
{
	double3 out;
	out.x = inx;
	out.y = iny;
	out.z = inz;
	return out;
}

// Function to convert float3 to double3
__device__ double3 float3_to_double3(float3 in)
{
	double3 out;
	out.x = in.x;
	out.y = in.y;
	out.z = in.z;
	return out;
}

// Function to add double3s
__device__ double3 operator+(double3 a, double3 b)
{
	return to_double3(a.x + b.x, a.y + b.y, a.z + b.z);
}

// Function to subtract double3s
__device__ double3 operator-(double3 a, double3 b)
{
	return to_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

// Function to divide double3 by double
__device__ double3 operator/(double3 a, double b)
{
	return to_double3(a.x/b, a.y/b, a.z/b);
}

// Function to compute length product of double3 and return it as a double
__device__ double lengthd3(double3 in)
{
	return sqrt(in.x*in.x + in.y*in.y + in.z*in.z);
}

// Function to get the squared magnitude of a double3
__device__ double magsquared3(double3 a)
{
	return (a.x*a.x + a.y*a.y + a.z*a.z);
}

// Function to normalise double3 input
__device__ double3 normalised3(double3 in)
{
	double norm = lengthd3(in);
	return to_double3(in.x/norm, in.y/norm, in.z/norm);
}

// Function to normalise float3 input
__device__ float3 normalise_float3(double in1, double in2, double in3)
{
	double norm = lengthd3(to_double3(in1, in2, in3));
	return make_float3(in1/norm, in2/norm, in3/norm);
}

// Function to compute dot product of two double3s
__device__ double dotd3(double3 a, double3 b)
{
	return a.x * b.x + a.y * b.y + a.z * b.z;;
}

// Function to convert double Cartesian coordinates to azimuth and elevation
__device__ double2 cart_to_sph(double3 in)
{
	double2 sph;
	sph.x = atan2(in.y, in.x);
	sph.y = atan2(in.z, sqrt(in.x*in.x + in.y*in.y));
	return sph;
}


/* Closest Hit Program */
RT_PROGRAM void closest_hit(void)
{
	// Print statement for debugging; does not print unless enabled in host code
	rtPrintf("Entering closest hit!\n\n");

	// If ray has not previously hit Earth or been received (i.e., ray has not been terminated)
	if ((prd.end == false) && ((prd.refrDepth < d_maxRefrDepth) || (prd.reflDepth < (d_maxReflDepth - 1)))) {

		// Find current ray's index
		unsigned int rayIndex = launchIndex.z*d_width*d_width + launchIndex.y*d_width + launchIndex.x;	// Ray launch index, excluding refractions

		// Add target index to ray path
		if (prd.refrDepth != 1) {					// Only do this for refrDepth of 0 or 2
			uint2 temp;								// Need uint2 to index 2D buffer
			temp.y = rayIndex + prd.maxRayIndex;	// Account for refractions too; y corresponds to height/row
			temp.x = prd.reflDepth + prd.refrDepth;	// x corresponds to width/column
			if (temp.x < (d_maxRefrDepth + d_maxReflDepth - 1))	
				dbuf_targ_intersect[temp] = (int)(d_targIndex);
		}

		// If a ray intersected a triangle (from intersection program), compute hit-point coordinate, ray length, and partial ray length; update PRD
		double3 hitPoint;
		hitPoint.x = (prd.prevHitPoint).x + (double)hit_t*(prd.rayDirection).x;
		hitPoint.y = (prd.prevHitPoint).y + (double)hit_t*(prd.rayDirection).y;
		hitPoint.z = (prd.prevHitPoint).z + (double)hit_t*(prd.rayDirection).z;
		prd.rayLength += hit_t;

		// if (rayIndex == 505063)
		// 	printf("[%.10e, %.10e, %.10e, %d];\n", (hitPoint).x, (hitPoint).y, (hitPoint).z, rayIndex);

		// Update PRD power and firstHitPoint
		if ((prd.reflDepth == 0) && (prd.refrDepth == 0)) {
			prd.firstHitPoint = hitPoint;							// Save first hit-point before any reflection or refraction; does not change again
			double3 TxRange = prd.firstHitPoint - d_rayOrigin;		// From Tx to first-hit target
			if (lengthd3(TxRange) >= SCENE_EPS)						// Must be larger than scene epsilon
				prd.power = 1/((magsquared3(TxRange))*4*M_PI);		// Set power with Tx (4*pi) and squared range from transmitter to first hit-point
			else
				prd.end = true;
		}
		else {
			double3 TargRange = hitPoint - prd.prevHitPoint;		// From previously-hit target to currently-hit target; PRD hit-point not yet updated
			if (lengthd3(TargRange) >= SCENE_EPS_R)					// Must be larger than scene epsilon for reflected rays
				prd.power *= 1/((magsquared3(TargRange))*4*M_PI);	// Update power with Target (4*pi) and squared range from previous target to current target
			else
				prd.end = true;
		}	// Note: If ray does not reflect/refract after this, it is nullified (see end of file) - so this else statement would not matter anyway

		// Update previous hit-point with current hit-point
		prd.prevHitPoint = hitPoint;

		// if ((prd.reflDepth == 0) && (prd.refrDepth == 0)) {
		// 	printf("[%e, %e, %e];\n", hitPoint.x, hitPoint.y, hitPoint.z);
		// }

		// Variables for refraction and reflection
		float3 hitPoint_f3 = make_float3(hitPoint.x, hitPoint.y, hitPoint.z);
		float3 new_direction;			// New direction for refracted ray, then reflected ray

		// Prepare for Doppler computation
		double3 V_targ = dbuf_targ_vel[d_targIndex];             		// Target velocity vector
		double3 k1, k0;													// k1 and k0 vectors (next and previous ray directions)

		// Backup the PRD for a refracted ray
		PerRayData prd_refr = prd;

		// Update the "previous" refraction index with the most recent value
		prd_refr.refrIndex.x = prd_refr.refrIndex.y;

		// If transmission coefficient would be zero, there is no need to compute refraction
		// Only refract on first intersection for this ray (no reflections yet)
		if ( (fabs(d_targReflCoeff) != 1.00000f) && (prd_refr.refrDepth < d_maxRefrDepth) && (prd_refr.reflDepth == 0))	// Use "<" since refrDepth is incremented INSIDE "if" statement
		{
			// Change refractive index ratio depending on current and previous medium of propagation
			if (prd_refr.refrIndex.x == 1) {									// If first medium is vacuum
				prd_refr.refrIndex.y = d_targRefrIndex;							// Second medium will be current target's material (refraction)
			}
			else {																// If first medium is current target's material
				prd_refr.refrIndex.y = 1;										// Second medium will be vacuum (refraction)
			}

			// Calculate index ratio; must be in float form
			float refr_index_ratio = (float)(prd_refr.refrIndex.y/prd_refr.refrIndex.x);			// Index ratio is n2 / n1 = current index / previous index

			// Check if ray actually will refract
			if ( refract( new_direction, ray.direction, normalise_float3(normal.x, normal.y, normal.z), refr_index_ratio ) )
			{
				unsigned int currentRayIndex = prd_refr.maxRayIndex + (d_width*d_width*d_width);	// Current ray index; write to this index of output buffer
				prd_refr.maxRayIndex = currentRayIndex;						// Increment ray index by total rays spawned when new refraction is created

				// Add currently-intersected target's index to this ray's "path chain"; ALWAYS copy ray's first refraction path to next refraction path
				// refrDepth will always be 0 (first hit) at this point; "currentRay" refers to NEXT refraction here
				// ONLY consider rays' FIRST target intersection, NOT subsequent reflections BEFORE refraction has occured for that ray
				// E.g. Ray 0 hits target, refracts; Ray 0 also reflects and hits another target but it will NOT refract there
				if ((prd_refr.refrDepth == 0) && (currentRayIndex == (d_width*d_width*d_width))) {
					uint2 temp;

					// First refracted ray, which will become "trapped" within the target
					for (unsigned int i = 0; i < (d_maxReflDepth + d_maxRefrDepth - 1); i++) {	// "Columns" of targ_intersect matrix
						temp.y = rayIndex + currentRayIndex;			// Index of refracted ray
						temp.x = i;										// Set all depth columns to current target index; "trapped" ray
						dbuf_targ_intersect[temp] = (int)(d_targIndex);	// "Trapped refracted ray" will forever hit the same target
					}

					// Subsequent refracted rays, which will eventually "exit" the target; start at j = 1 to skip first (above) refracted ray
					for (unsigned int j = 0; j < d_maxReflDepth; j++) {		// Any refraction will give rise to d_maxReflDepth refracted rays (total)
						for (unsigned int i = 0; i < (j + 2); i++) {		// "Columns" of targ_intersect matrix
							temp.y = rayIndex + (j + 2)*currentRayIndex;	// Index of refracted ray
							temp.x = i;										// Set relevant depth columns to current target index
							dbuf_targ_intersect[temp] = (int)(d_targIndex);
						}
					}
				}

				// Create new refracted ray; use INCIDENT scene epsilon
				Ray refr_ray = make_Ray( hitPoint_f3, new_direction, 0, SCENE_EPS, RT_DEFAULT_MAX );

				// Update current PRD for refraction; if max reflDepth reached, ALL ray power is transferred to refracted ray (i.e. prd_refr.power *= 1)
				if ((prd_refr.reflDepth + 1) < d_maxReflDepth)			// If maximum reflection depth is not yet being reached with this intersection (incremented later)
					prd_refr.power *= (1 - fabs(d_targReflCoeff));	// Update refracted ray's power with "power loss" (transmission/reflection)
				prd_refr.refrDepth++;

				// Calculate Doppler shift for this target using k1, k0 and V_targ; (k1 - k0) can be minimum of -2 or maximum of +2 (unit vectors)
				// (k1 - k0) corresponds to the term 2cos(B/2) in the bistatic Fd equation
				k0 = normalised3(prd_refr.rayDirection); 					// Normalised k0 (current ray direction) using double3 version
				prd_refr.rayDirection = float3_to_double3(new_direction);	// Update double3 version of ray direction in PRD
				k1 = normalised3(prd_refr.rayDirection);					// Normalised k1 (next ray direction) using double3 version

				// Add target Doppler velocity to running total (Battaglia, 2011); uses (k1 - k0) to account for negative Fd away from radar
				prd_refr.doppler += dotd3(V_targ, (k1 - k0));	// For refractions, this may be zero since k1 could be the same as k0

				// Save refracted RCS angles (for FileTargets)
				uint2 temp_rcs;											// Need uint2 to index 2D buffer
				temp_rcs.y = rayIndex + currentRayIndex;				// Account for refractions too; y corresponds to height/row; overall ray index
				temp_rcs.x = prd_refr.reflDepth + (prd_refr.refrDepth - 1);		// x corresponds to width/column; -1 to "replicate" targ_intersect uint2 above
				double2 k0_sph = cart_to_sph(k0);								// From previous point to current point (as per FERS)
				double2 k1_sph = cart_to_sph(to_double3(-k1.x, -k1.y, -k1.z));	// Reverse direction so that -k1 goes from next point to current point (as per FERS)
				dbuf_rcs_angle[temp_rcs].x = k0_sph.x + k1_sph.x;		// tAngle azi
				dbuf_rcs_angle[temp_rcs].y = k0_sph.y + k1_sph.y;		// tAngle ele
				
				// Recursively call rtTrace for refracted ray
				rtTrace(d_targets_all, refr_ray, prd_refr);

				// When refraction rtTrace finishes ray traversal, save results; for reflected rays, this is done at the end of the ray generation program
				// Need to use currentRayIndex, NOT maxRayIndex since that accounts for all (nested) refracted ray indices
				dbuf_results[rayIndex + currentRayIndex].reflDepth = prd_refr.reflDepth;
				dbuf_results[rayIndex + currentRayIndex].refrDepth = prd_refr.refrDepth;
				dbuf_results[rayIndex + currentRayIndex].rayLength = prd_refr.rayLength;
				dbuf_results[rayIndex + currentRayIndex].firstHitPoint = prd_refr.firstHitPoint;
				dbuf_results[rayIndex + currentRayIndex].prevHitPoint = prd_refr.prevHitPoint;
				dbuf_results[rayIndex + currentRayIndex].power = prd_refr.power;
				dbuf_results[rayIndex + currentRayIndex].doppler = prd_refr.doppler;
				dbuf_results[rayIndex + currentRayIndex].received = prd_refr.received;
				// dbuf_results[rayIndex + currentRayIndex].rayDirection = prd_refr.rayDirection;	// REMOVE
			}
		}

		/// AFTER THE REFRACTIONS HAVE RECURSIVELY COMPLETED PROCESSING
		// Increment reflDepth OUTSIDE/BEFORE "if" statement; essentially counts the number of intersections of each ray
		prd.reflDepth++;

		// Reflected ray will ALWAYS propagate through the "previous" medium before intersection; affects the next reflection/refraction
		prd.refrIndex.y = prd_refr.refrIndex.x;
		prd.refrIndex.x = prd_refr.refrIndex.x;

		// If the number of ray bounces is below "stop index", then recursively call rtTrace to continue ray traversal
		if (prd.reflDepth < d_maxReflDepth) {	// Use "<" so that d_maxReflDepth is the "stop index" at which reflections are stopped; max. reflections per ray = (d_maxReflDepth - 1)
			
			// Compute reflected ray
			new_direction = reflect( ray.direction, normalise_float3(normal.x, normal.y, normal.z) );	// Must be float3 for OptiX function
			Ray refl_ray = make_Ray( hitPoint_f3, new_direction, 0, SCENE_EPS_R, RT_DEFAULT_MAX );
			prd.power *= d_targReflCoeff;		// Update reflected ray's power with "power loss" (transmission/reflection)

			// Calculate Doppler shift for this target using k1, k0 and V_targ; (k1 - k0) can be minimum of -2 or maximum of +2 (unit vectors)
			// (k1 - k0) corresponds to the term 2cos(B/2) in the bistatic Fd equation
			k0 = normalised3(prd.rayDirection); 					// Normalised k0 (current ray direction) using double3 version
			prd.rayDirection = float3_to_double3(new_direction);	// Update double3 version of ray direction in PRD
			k1 = normalised3(prd.rayDirection);						// Normalised k1 (next ray direction) using double3 version

			// Add target Doppler velocity to running total (Battaglia, 2011); uses (k1 - k0) to account for negative Fd away from radar
			// // CHANGE: Test case
			// if (prd.refrDepth > 0) {
			// 	double3 V_targ2;
			// 	V_targ2.x = V_targ.x * 2; V_targ2.y = V_targ.y * 2; V_targ2.z = V_targ.z * 2;
			// 	prd.doppler += dotd3(V_targ2, (k1 - k0));
			// }
			// else
				prd.doppler += dotd3(V_targ, (k1 - k0));
			
			// if (rayIndex == 504910)
			// 	printf("%e\n", prd.doppler);

			// Save reflected RCS angles (for FileTargets)
			uint2 temp_rcs;											// Need uint2 to index 2D buffer
			temp_rcs.y = rayIndex + prd.maxRayIndex;				// maxRayIndex tracks the "refraction set" to which the ray belongs, e.g. ray 0 means maxRayIndex = 0, ray 1000 means maxRayIndex = 1000
			temp_rcs.x = (prd.reflDepth - 1) + prd.refrDepth;				// x corresponds to width/column; -1 to "replicate" targ_intersect uint2 above
			double2 k0_sph = cart_to_sph(k0);								// From previous point to current point (as per FERS)
			double2 k1_sph = cart_to_sph(to_double3(-k1.x, -k1.y, -k1.z));	// Reverse direction so that -k1 goes from next point to current point (as per FERS)
			dbuf_rcs_angle[temp_rcs].x = k0_sph.x + k1_sph.x;		// tAngle azi
			dbuf_rcs_angle[temp_rcs].y = k0_sph.y + k1_sph.y;		// tAngle ele

			// if (prd.reflDepth > 0)
			// 	printf("[%e, %e, %e, %d];\n", new_direction.x, new_direction.y, new_direction.z, prd.reflDepth);

			// Recursively call rtTrace for reflected ray
			rtTrace(d_targets_all, refl_ray, prd);			// Recursively call rtTrace for reflected ray
		}

		// If number of ray reflections and refractions have exceeded the maximum numbers allowed
		if ((prd.reflDepth + 1 >= d_maxReflDepth) && (prd.refrDepth >= d_maxRefrDepth)) {
			prd.end = true;	// Stop ray from being received; treat ray as if its energy is absorbed by the object it just hit
		}
	}
}
