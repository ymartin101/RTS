#include "hip/hip_runtime.h"

/* ****************** Host & Device Code ****************** 

* Error checker
* Device code
* Computes ray aggregation (power scaling)
* Host code
* Set up code for kernel launch

************************************************ */

#include "aggregation.cuh"

/* *************** ERROR CHECKER *************** */

// Check for CUDA errors
#define cudaCheckErrors(msg) \
	do { \
		hipError_t __err = hipGetLastError(); \
		if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
				msg, hipGetErrorString(__err), \
				__FILE__, __LINE__); \
			fprintf(stderr, "*** FAILED - ABORTING\n\n"); \
			exit(1); \
		} \
	} while (0)


/* *************** DEVICE CODE: RAY AGGREGATION KERNEL 1 *************** */
					
extern "C" __global__ void myKernel1(PerRayData* d_results_arr, int* d_targ_intersect_arr, unsigned int receivedRays, \
	unsigned int depthTotal, unsigned int MaxThreads, unsigned int MaxBlocks, double* d_npath_arr, \
	double* d_power_arr, double* d_doppler_arr, double* d_delay_arr, double* d_phase_arr, double cspeed, \
	double carrier, int* d_pathMatch)
{
	// Only use threads up until the end of receivedRays
	int tid = int(threadIdx.x + (blockIdx.x * blockDim.x));	// Thread index
	int stride = blockDim.x * gridDim.x;  						// Total number of threads spawned
	for (int i = tid; i < receivedRays; i += stride){			// Will only iterate again if receivedRays is more than (MaxBlocks*MaxThreads)

		// Iterate through all RECEIVED rays and compare to ray i
		for (unsigned int r = 0; r < receivedRays; r++) {

			// FIRST check for same Rx (to cut down on computations)
			if (d_results_arr[i].received == d_results_arr[r].received) {
					bool row_equal = true;
					for(unsigned int k = 0; k < depthTotal; k++) { // Find ray path for ray r
						if (d_targ_intersect_arr[k + i*depthTotal] != d_targ_intersect_arr[k + r*depthTotal]) {
							row_equal = false;	// Rows are not equal, so immediately abandon this loop
							break;
						}
					}

				// Check/compare paths for ray i and ray r (and check for direct transmission)
				if ((row_equal == true) || ((d_results_arr[i].reflDepth == 0) && (d_results_arr[i].refrDepth == 0))) {
						
					// Update ray quantity totals
					double delay = (d_results_arr[r].rayLength)/cspeed;
					double phase = -fmod(delay*2*M_PI*carrier, 2*M_PI);
					d_npath_arr[i] += 1;											// Increment for matching paths
					d_power_arr[i] += sqrt(d_results_arr[r].power);					// Add "voltages" for common-path rays
					d_delay_arr[i] += delay;										// Add delays for common-path rays
					d_phase_arr[i] += phase;										// Add phases for common-path rays
					d_doppler_arr[i] += d_results_arr[r].doppler;					// Add Dopplers for common-path rays
					
					// Record current ray index as this path's "path index"
					if (r < d_pathMatch[i])		// Always record the "earliest" ray index that follows this path
						d_pathMatch[i] = r;		// Note: iterating through rays "r" will include ray "i" as well
				}
			}
		}
	}
}
 

/* *************** DEVICE CODE: RAY AGGREGATION KERNEL 2 *************** */
					
extern "C" __global__ void myKernel2(PerRayData* d_results_arr, unsigned int receivedRays, double* d_npath_arr, \
	double* d_power_arr, double* d_doppler_arr, double* d_delay_arr, double* d_phase_arr)
{
	// Only use threads up until the end of receivedRays
	int tid = int(threadIdx.x + (blockIdx.x * blockDim.x));	// Thread index
	int stride = blockDim.x * gridDim.x;  						// Total number of threads spawned
	for (int i = tid; i < receivedRays; i += stride){			// Will only iterate again if receivedRays is more than (MaxBlocks*MaxThreads)

		// Divide totals of quantities by Npath (for same-path rays); done in Kernel 2 to avoid hbuf etc being overwritten during Kernel 1 processing
		if (d_npath_arr[i] > 0) {
			d_results_arr[i].power = pow(d_power_arr[i]/d_npath_arr[i], 2);	// Divide total voltage by Npath (averaging), then square
			d_delay_arr[i] /= d_npath_arr[i];									// Divide total delay by Npath
			d_phase_arr[i] /= d_npath_arr[i];									// Divide total phase by Npath
			d_results_arr[i].doppler = d_doppler_arr[i]/d_npath_arr[i];		// Divide total Doppler by Npath
			
			// printf("Power: %e, Npath: %lf\n", d_results_arr[i].power, d_npath_arr[i]);
		}
	}
}


/* *************** HOST CODE: RAY AGGREGATION SETUP *************** */

namespace rs {
	void kernel_wrapper(PerRayData* h_rx_results_arr, int* h_rx_intersects_arr, unsigned int receivedRays, \
		unsigned int depthTotal, unsigned int MaxThreads, unsigned int MaxBlocks, double cspeed, double carrier, \
		double* h_npath_arr, double* h_power_arr, double* h_doppler_arr, double* h_delay_arr, double* h_phase_arr, \
		int* h_pathMatch)
	{ 	   
		// Variables to send to device
		PerRayData* d_results_arr;
		int* d_targ_intersect_arr;
		double* d_npath_arr; double* d_power_arr; double* d_doppler_arr; double* d_delay_arr; double* d_phase_arr;
		int* d_pathMatch;	// Tracks path matches

		// Allocate memory on the device
		hipMalloc((void **)&d_results_arr, sizeof(PerRayData)*receivedRays);
		hipMalloc((void **)&d_targ_intersect_arr, sizeof(int)*receivedRays*depthTotal);
		hipMalloc((void **)&d_power_arr, sizeof(double)*receivedRays);
		hipMalloc((void **)&d_doppler_arr, sizeof(double)*receivedRays);
		hipMalloc((void **)&d_delay_arr, sizeof(double)*receivedRays);
		hipMalloc((void **)&d_phase_arr, sizeof(double)*receivedRays);
		hipMalloc((void **)&d_npath_arr, sizeof(double)*receivedRays);
		hipMalloc((void **)&d_pathMatch, sizeof(int)*receivedRays);
		cudaCheckErrors("Malloc fail");

		// Copy inputs to device
		hipMemcpy(d_results_arr, h_rx_results_arr, sizeof(PerRayData)*receivedRays, hipMemcpyHostToDevice);
		hipMemcpy(d_targ_intersect_arr, h_rx_intersects_arr, sizeof(int)*receivedRays*depthTotal, hipMemcpyHostToDevice);
		hipMemcpy(d_npath_arr, h_npath_arr, sizeof(double)*receivedRays, hipMemcpyHostToDevice);
		hipMemcpy(d_power_arr, h_power_arr, sizeof(double)*receivedRays, hipMemcpyHostToDevice);
		hipMemcpy(d_doppler_arr, h_doppler_arr, sizeof(double)*receivedRays, hipMemcpyHostToDevice);
		hipMemcpy(d_delay_arr, h_delay_arr, sizeof(double)*receivedRays, hipMemcpyHostToDevice);
		hipMemcpy(d_phase_arr, h_phase_arr, sizeof(double)*receivedRays, hipMemcpyHostToDevice);
		hipMemcpy(d_pathMatch, h_pathMatch, sizeof(int)*receivedRays, hipMemcpyHostToDevice);
		cudaCheckErrors("Memory (device) fail");

		// Timer for ray aggregation kernels runtime
		struct timeval timer4;
		gettimeofday(&timer4, NULL);
		double StartTime_RA = timer4.tv_sec + (timer4.tv_usec/1000000.0);

		// Ray aggregation kernel 1 launch
		if (receivedRays <= MaxThreads)				// If there are fewer rays than the number of threads in one block (or an equal number of targets)
			myKernel1<<<1, receivedRays>>>(d_results_arr, d_targ_intersect_arr, receivedRays, depthTotal, MaxThreads, \
				MaxBlocks, d_npath_arr, d_power_arr, d_doppler_arr, d_delay_arr, d_phase_arr, cspeed, carrier, d_pathMatch);
		else if (receivedRays > (MaxThreads*MaxBlocks))	// If there are more rays than the maximum number of parallel threads (across all blocks)
			myKernel1<<<MaxBlocks, MaxThreads>>>(d_results_arr, d_targ_intersect_arr, receivedRays, depthTotal, MaxThreads, \
				MaxBlocks, d_npath_arr, d_power_arr, d_doppler_arr, d_delay_arr, d_phase_arr, cspeed, carrier, d_pathMatch);
		else										// If number of rays requires more than 1 block, but not all of them
			myKernel1<<<((receivedRays + (MaxThreads - 1))/MaxThreads), MaxThreads>>>(d_results_arr, d_targ_intersect_arr, \
				receivedRays, depthTotal, MaxThreads, MaxBlocks, d_npath_arr, \
				d_power_arr, d_doppler_arr, d_delay_arr, d_phase_arr, cspeed, carrier, d_pathMatch);
		cudaCheckErrors("Kernel 1 fail");

		// Ray aggregation kernel 2 launch
		if (receivedRays <= MaxThreads)				// If there are fewer rays than the number of threads in one block (or an equal number of targets)
			myKernel2<<<1, receivedRays>>>(d_results_arr, receivedRays, d_npath_arr, d_power_arr, d_doppler_arr, d_delay_arr, d_phase_arr);
		else if (receivedRays > (MaxThreads*MaxBlocks))	// If there are more rays than the maximum number of parallel threads (across all blocks)
			myKernel2<<<MaxBlocks, MaxThreads>>>(d_results_arr, receivedRays, d_npath_arr, d_power_arr, d_doppler_arr, d_delay_arr, d_phase_arr);
		else										// If number of rays requires more than 1 block, but not all of them
			myKernel2<<<((receivedRays + (MaxThreads - 1))/MaxThreads), MaxThreads>>>(d_results_arr, receivedRays, d_npath_arr, d_power_arr, d_doppler_arr, d_delay_arr, d_phase_arr);
		cudaCheckErrors("Kernel 2 fail");

		// Timer for ray aggregation runtime
        gettimeofday(&timer4, NULL);
        double RTS_RA_time = timer4.tv_sec + (timer4.tv_usec/1000000.0) - StartTime_RA;
        printf("Ray aggregation took %lf seconds.\n", RTS_RA_time);

		// Copy from device 
		hipMemcpy(h_rx_results_arr, d_results_arr, sizeof(PerRayData)*receivedRays, hipMemcpyDeviceToHost);
		hipMemcpy(h_delay_arr, d_delay_arr, sizeof(double)*receivedRays, hipMemcpyDeviceToHost);
		hipMemcpy(h_phase_arr, d_phase_arr, sizeof(double)*receivedRays, hipMemcpyDeviceToHost);
		hipMemcpy(h_pathMatch, d_pathMatch, sizeof(int)*receivedRays, hipMemcpyDeviceToHost);
		cudaCheckErrors("Memory (host) fail");

		// Free memory
		hipFree(d_results_arr);
		hipFree(d_targ_intersect_arr);
		hipFree(d_delay_arr);
		hipFree(d_phase_arr);
		hipFree(d_npath_arr);
		cudaCheckErrors("Delete fail");

		return;
	}
}
